#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cuda_util.h"
#include <array>
#include <functional>
#include <iostream>
#include <memory>

constexpr int M{4096};
constexpr int N{4096};
constexpr int NELEMENTS{M * N};

float measure_performance(std::function<void(hipStream_t)> bound_function,
                          hipStream_t stream, int num_repeats = 1000,
                          int num_warmups = 100) {
  hipEvent_t start, stop;
  float time;

  CHECK_CUDA_ERROR(hipEventCreate(&start));
  CHECK_CUDA_ERROR(hipEventCreate(&stop));

  for (int i{0}; i < num_warmups; ++i) {
    bound_function(stream);
  }

  CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

  CHECK_CUDA_ERROR(hipEventRecord(start, stream));
  for (int i{0}; i < num_repeats; ++i) {
    bound_function(stream);
  }
  CHECK_CUDA_ERROR(hipEventRecord(stop, stream));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop));
  CHECK_LAST_CUDA_ERROR();
  CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
  CHECK_CUDA_ERROR(hipEventDestroy(start));
  CHECK_CUDA_ERROR(hipEventDestroy(stop));

  const float latency{time / num_repeats};

  return latency;
}

__global__ void copy_v1(float *src, float *dst) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int in_idx = y * M + x;
  dst[in_idx] = src[in_idx];
}

void copy_v1_test(float *d_src, float *d_dst, hipStream_t stream) {
  dim3 grid(M / 32, N / 32);
  dim3 block(32, 32);
  copy_v1<<<grid, block, 0, stream>>>(d_src, d_dst);
}

__global__ void copy_v2(float *src, float *dst) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx] = src[idx];
}

void copy_v2_test(float *d_src, float *d_dst, hipStream_t stream) {
  dim3 grid(M * N / 256);
  dim3 block(256);
  copy_v2<<<grid, block, 0, stream>>>(d_src, d_dst);
}

// Note: naive implementation
__global__ void transpose_v1(float *src, float *dst) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int in_idx = y * M + x;
  int out_idx = x * N + y;
  dst[out_idx] = src[in_idx];
}

void transpose_v1_test(float *d_src, float *d_dst, hipStream_t stream) {
  dim3 grid(M / 32, N / 32);
  dim3 block(32, 32);
  transpose_v1<<<grid, block, 0, stream>>>(d_src, d_dst);
}

// Note: share memory implementation w/ bank conflict
__global__ void transpose_v2(float *src, float *dst) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int in_idx = y * M + x;
  __shared__ float buf[32][32];
  buf[threadIdx.y][threadIdx.x] = src[in_idx];
  __syncthreads();
  int xx = blockIdx.y * blockDim.y + threadIdx.x;
  int yy = blockIdx.x * blockDim.x + threadIdx.y;
  dst[yy * M + xx] = buf[threadIdx.x][threadIdx.y];
}

void transpose_v2_test(float *d_src, float *d_dst, hipStream_t stream) {
  dim3 grid(M / 32, N / 32);
  dim3 block(32, 32);
  transpose_v2<<<grid, block, 0, stream>>>(d_src, d_dst);
}

// Note: share memory implementation w/o bank conflict
__global__ void transpose_v3(float *src, float *dst) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int in_idx = y * M + x;
  __shared__ float buf[32][33];
  buf[threadIdx.y][threadIdx.x] = src[in_idx];
  __syncthreads();
  int xx = blockIdx.y * blockDim.y + threadIdx.x;
  int yy = blockIdx.x * blockDim.x + threadIdx.y;
  dst[yy * M + xx] = buf[threadIdx.x][threadIdx.y];
}

void transpose_v3_test(float *d_src, float *d_dst, hipStream_t stream) {
  dim3 grid(M / 32, N / 32);
  dim3 block(32, 32);
  transpose_v3<<<grid, block, 0, stream>>>(d_src, d_dst);
}

bool all_close(float *h_src, float *h_dst, float eps = 1e-5) {
  for (int i{0}; i < M; ++i) {
    for (int j{0}; j < N; ++j) {
      if (std::abs(h_src[i * N + j] - h_dst[j * M + i]) > eps) {
        return false;
      }
    }
  }
  return true;
}

int main() {
  float *h_src, *h_dst, *d_src, *d_dst;
  CHECK_CUDA_ERROR(hipHostMalloc(&h_src, sizeof(float) * NELEMENTS));
  CHECK_CUDA_ERROR(hipHostMalloc(&h_dst, sizeof(float) * NELEMENTS));
  CHECK_CUDA_ERROR(hipMalloc(&d_src, sizeof(float) * NELEMENTS));
  CHECK_CUDA_ERROR(hipMalloc(&d_dst, sizeof(float) * NELEMENTS));
  for (int i{0}; i < M; ++i) {
    for (int j{0}; j < N; ++j) {
      h_src[i * N + j] = i * N + j;
    }
  }
  CHECK_CUDA_ERROR(hipMemcpy(d_src, h_src, sizeof(float) * NELEMENTS,
                              hipMemcpyHostToDevice));
  hipStream_t stream;
  CHECK_CUDA_ERROR(hipStreamCreate(&stream));

  {
    std::function<void(hipStream_t)> copy_v1_test_warp{
        std::bind(copy_v1_test, d_src, d_dst, std::placeholders::_1)};
    auto latency = measure_performance(copy_v1_test_warp, stream);
    float bandwidth = M * N * sizeof(float) * 2 / latency * 1e-6;
    std::cout << "Bandwidth: " << bandwidth << "GB/s.\n";
  }

  {
    std::function<void(hipStream_t)> copy_v2_test_warp{
        std::bind(copy_v2_test, d_src, d_dst, std::placeholders::_1)};
    auto latency = measure_performance(copy_v2_test_warp, stream);
    float bandwidth = M * N * sizeof(float) * 2 / latency * 1e-6;
    std::cout << "Bandwidth: " << bandwidth << "GB/s.\n";
  }

  {
    std::function<void(hipStream_t)> transpose_v1_test_warp{
        std::bind(transpose_v1_test, d_src, d_dst, std::placeholders::_1)};
    auto latency = measure_performance(transpose_v1_test_warp, stream);
    float bandwidth = M * N * sizeof(float) * 2 / latency * 1e-6;
    std::cout << "Bandwidth: " << bandwidth << "GB/s.\n";
  }

  {
    std::function<void(hipStream_t)> transpose_v2_test_warp{
        std::bind(transpose_v2_test, d_src, d_dst, std::placeholders::_1)};
    auto latency = measure_performance(transpose_v2_test_warp, stream);
    float bandwidth = M * N * sizeof(float) * 2 / latency * 1e-6;
    std::cout << "Bandwidth: " << bandwidth << "GB/s.\n";
  }

  {
    std::function<void(hipStream_t)> transpose_v3_test_warp{
        std::bind(transpose_v3_test, d_src, d_dst, std::placeholders::_1)};
    auto latency = measure_performance(transpose_v3_test_warp, stream);
    float bandwidth = M * N * sizeof(float) * 2 / latency * 1e-6;
    std::cout << "Bandwidth: " << bandwidth << "GB/s.\n";
  }

  CHECK_CUDA_ERROR(hipMemcpy(h_dst, d_dst, sizeof(float) * NELEMENTS,
                              hipMemcpyDeviceToHost));
  if (all_close(h_src, h_dst)) {
    std::cout << "Success.\n";
  } else {
    std::cout << "Failed.\n";
  }

  CHECK_CUDA_ERROR(hipHostFree(h_src));
  CHECK_CUDA_ERROR(hipHostFree(h_dst));
  CHECK_CUDA_ERROR(hipFree(d_src));
  CHECK_CUDA_ERROR(hipFree(d_dst));
  CHECK_CUDA_ERROR(hipStreamDestroy(stream));
}
